/**************************************
***************************************
* Code Can be compiled using --> nvcc kernel5.cu -lcurand if the cuRand lib is the envirement PATH
* else use nvcc kernel5.cu -L</path/to/the/lib> -lcurand 
***************************************
**************************************/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>

__global__ void MC_test(unsigned int seed,hiprandState *states,unsigned int numsim,unsigned int *results)
{
    extern __shared__ int sdata[];
    int i;
    int nthreads = gridDim.x * blockDim.x;
    unsigned int innerpoint=0;
    int tx=threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tx;
    hiprandState *state =states + idx;
    float x,y,l2norm2;
    sdata[tx]=0;
    __syncthreads();
    hiprand_init(seed, tx, 0, state);
    __syncthreads();
    for(i=tx;i<numsim;i+=nthreads){
         x = hiprand_uniform(state);
         y = hiprand_uniform(state);
         l2norm2 = x * x + y * y;
        if (l2norm2 < static_cast<float>(1))
            {
                innerpoint++;;
            }
    }   
    __syncthreads();
    sdata[tx]=innerpoint;
    __syncthreads();
    //-------reduction
    for (unsigned int s=blockDim.x/2;s>0;s>>=1){
        if(tx < s){
            sdata[tx]=sdata[tx]+sdata[tx+s];
        }
    }
    //-----------------
    __syncthreads();   
    if(tx==0){
    results[blockIdx.x]=sdata[0];
    }
   
}
void caller(unsigned int numsim){
    static hiprandState *states=NULL;
    unsigned int *results;
    unsigned int seed=rand();
    float pi=0;
    float r_pi= 3.14159265358979323846;
    dim3 block;
    dim3 grid;
    block.x=1<<10;
    grid.x=2;//=(numsim +block.x -1)/block.x; //ceil((float)numsim/(float)(block.x));
    printf(" \n grid %d block %d  ",grid.x,block.x);
    hipMallocManaged(&states,sizeof(hiprandState)*block.x * grid.x);
    hipMallocManaged(&results,2*sizeof(unsigned int));
    results[0]=0;
    results[1]=0;
    MC_test<<<grid , block, block.x*sizeof(unsigned int)>>>(seed,states,numsim,results);
    hipDeviceSynchronize();
    pi=4*(float)(results[0]+results[1])/(float)(numsim);
    printf(":: sims= %d, MC_pi= %f , error= %f  \t",numsim,pi,abs(pi-r_pi));
    hipFree(states);
}

int main(){
 unsigned int N=50;
 for (int i=1; i < N ;i++){
    caller(1<<i);
 }
 printf("\n");
    return 0;
}






